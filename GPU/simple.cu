#include <stdio.h>

/**
 * Source: https://developer.nvidia.com/content/how-query-device-properties-and-handle-errors-cuda-cc
 * Documentation of hipDeviceProp_t: http://docs.nvidia.com/cuda/cuda-runtime-api/index.html#structcudaDeviceProp
 *
 * Sample line to compile:
 *   nvcc -arch=sm_20 -o printDeviceInfo printDeviceInfo.cu
 */

int main() {
        int nDevices;

        hipGetDeviceCount(&nDevices);
        for (int i = 0; i < nDevices; i++) {
                hipDeviceProp_t prop;
                hipGetDeviceProperties(&prop, i);
                printf("Device Number: %d\n", i);
                printf("  Device name: %s\n", prop.name);
                printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
                printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
                printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        }
}
